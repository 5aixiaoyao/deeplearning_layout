
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cassert>
#include <math.h>

#define THREADS_PER_BLOCK 256
#define WARP_SIZE 32
#define DIVUP(m, n) ((m + n - 1) / n) 
// #define PI 3.1415926

__global__ void grid2points_kernel(float* points, int* indices, float* grid, int n, int n_channels){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// int tid = threadIdx.x;
	if (idx >= n){
		return ;
	}
    for (int i_channels = 0; i_channels < n_channels; i_channels++){
        points[idx * n_channels + i_channels] = grid[indices[idx] * n_channels + i_channels];
        // printf("points[idx + i_channels]: %f\n", grid[indices[idx] * n_channels + i_channels]);
    }
	return ;
}
void grid2points_launcher(float* points, int* indices, float* grid, int n, int n_channels){
	dim3 blockSize(DIVUP(n, THREADS_PER_BLOCK));
	dim3 threadSize(THREADS_PER_BLOCK);
	grid2points_kernel<<<blockSize, threadSize>>>(points, indices, grid, n, n_channels);
}
